
#include <hip/hip_runtime.h>
#include "math.h"
#define SMALLEST_FLOAT 1.175494351E-38
#define MAX_ELEMENTS_PER_BLOCK 2048
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n)\
		((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n)((n) >> LOG_NUM_BANKS)
#endif

extern "C"
__global__ void reduceStates(int numCategories, int numStates, int numSitesWithPadding, int numSites,
                    double pInv, double* equiFreq,
                    float* sequence, double* ufScaling){
    

    //among-site rate heterogenity category index
    int categoryIdx = blockIdx.y;
	
    // sequence site index
    int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	// site index within current block
	int siteBlockIdx = threadIdx.x; 
    // index of the ancestral state at the current node
    int stateIdx = threadIdx.y;



	__shared__ float partialLikelihoods[1024];
	__shared__ double sharedDoubleBuffer[1024];
	
	// Prefetching partial likelihoods
	if(siteIdx < numSitesWithPadding){
		partialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = sequence[siteIdx + stateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
	}
	
	
	if(siteIdx < numSites){	
		sharedDoubleBuffer[stateIdx * blockDim.x + siteBlockIdx] = partialLikelihoods[stateIdx * blockDim.x + siteBlockIdx] * equiFreq[stateIdx]
											* ((1.0 - pInv)/(double)(numCategories)) * ufScaling[categoryIdx * numSitesWithPadding + siteIdx];
	}
    
	__syncthreads();
	
	// Reduce states here
	if(stateIdx == 0 && siteIdx < numSites){
		double cellLikelihoodAccumulator = 0;
		for(int st = 0; st < numStates; st++){
			cellLikelihoodAccumulator += sharedDoubleBuffer[st * blockDim.x + siteBlockIdx];
		}
		ufScaling[categoryIdx * numSitesWithPadding + siteIdx] = cellLikelihoodAccumulator;
	}
}

extern "C"
__global__ void reduceCategories(int numCategories, int numSites, int numSitesWithPadding, double pInv, double* ufScaling, double* invSites, double* weights){
	int categoryIdx = threadIdx.y;
	int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int siteBlockIdx = threadIdx.x;
	
	__shared__ double sharedDoubleBuffer[1024];
	
	if(siteIdx < numSites){
		sharedDoubleBuffer[categoryIdx * blockDim.x + siteBlockIdx] = ufScaling[categoryIdx * numSitesWithPadding + siteIdx];
	}
	
	__syncthreads();
	
	if(categoryIdx == 0 && siteIdx < numSites){
		for(int cat = 1; cat < numCategories; cat++){
			sharedDoubleBuffer[siteBlockIdx] += sharedDoubleBuffer[cat * blockDim.x + siteBlockIdx];
		}
		double siteLikelihoodInv = invSites[siteIdx] * pInv;
		ufScaling[siteIdx] = log(sharedDoubleBuffer[siteBlockIdx] + siteLikelihoodInv) * weights[siteIdx];
	}
	
}

extern "C"
__global__ void reduceSites(double* g_odata, double* g_idata, int n, double* debug){
	
	__shared__ double temp[2115];
	int thid = threadIdx.x;
	int offset = 1;
	
	int ai = thid;  
	int bi = thid + (MAX_ELEMENTS_PER_BLOCK/2);  
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);  
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	//Copy data from global memory to shared memory and apply
	// padding for sizes that are not exponents of 2
	
	int blockOffset = MAX_ELEMENTS_PER_BLOCK * blockIdx.x;
	
	if((blockOffset + ai) < n){
		temp[ai + bankOffsetA] = g_idata[blockOffset + ai];
	}else{
		temp[ai + bankOffsetA] = 0;
	}
	
	if((blockOffset + bi) < n){
		temp[bi + bankOffsetB] = g_idata[blockOffset + bi];
	}else{
		temp[bi + bankOffsetB] = 0;
	}
	
	for(int d = MAX_ELEMENTS_PER_BLOCK >> 1; d > 0; d >>= 1){
		__syncthreads();
		if(thid < d){
			int ai = offset * (2 * thid + 1) - 1;
			int bi = offset * (2 * thid + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	
	if(thid == 0){
		g_idata[blockIdx.x] = temp[MAX_ELEMENTS_PER_BLOCK - 1 + CONFLICT_FREE_OFFSET(MAX_ELEMENTS_PER_BLOCK - 1)];
	}	
	__syncthreads();
	if(thid == 0 && blockIdx.x == 0 && gridDim.x == 1) g_odata[0] = g_idata[0];
}