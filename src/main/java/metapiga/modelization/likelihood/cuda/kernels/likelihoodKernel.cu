
#include <hip/hip_runtime.h>
#include "math.h"
#define SMALLEST_FLOAT 1.175494351E-38
#define MAX_ELEMENTS_PER_BLOCK 2048
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n)\
		((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n)((n) >> LOG_NUM_BANKS)
#endif

extern "C"
__global__ void JukesCantorGpu(int numCategories, int numStates, int numSitesWithPadding, int numSites,
                    double apRate, double* rates, double rateScaling, 
                    float* seqNodeLeft, float* seqNodeRight, float* seqAncNode,
                    double blLeft, double blRight,
                    double* ufScaling){
    
    
    //among-site rate heterogenity category index
    int categoryIdx = blockIdx.y;
	
    // sequence site index
    int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	// site index within current block
	int siteBlockIdx = threadIdx.x;
    // index of the ancestral state at the current node
    int ancStateIdx = threadIdx.y;



	__shared__ double ancSequenceSharedRecord[1024];
	__shared__ float leftPartialLikelihoods[1024];
	__shared__ float rightPartialLikelihoods[1024];
	__shared__ double ufScalingSharedMem[512];
	//__shared__ float parentPartialLikelihoods[1024];
	
	// Prefetching partial likelihoods
	if(siteIdx < numSitesWithPadding){
		leftPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeLeft[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
		rightPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeRight[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
	}

    double categoryRate = rates[categoryIdx];
	
	double expLeft = exp(-blLeft * rateScaling * categoryRate * apRate);
    double diagLeft  = 0.25 + (0.75 * expLeft);
    double offdiagLeft = 0.25 - (0.25 * expLeft);

    double expRight = exp(-blRight * rateScaling * categoryRate * apRate);
    double diagRight  = 0.25 + (0.75 * expRight);
    double offdiagRight = 0.25 - (0.25 * expRight);

    int ancSequenceIdx = categoryIdx * numStates * numSitesWithPadding + ancStateIdx * numSitesWithPadding + siteIdx;

	double sumLeft = 0;
	double sumRight = 0;
	if(siteIdx < numSitesWithPadding){

		for(int descStateIdx = 0; descStateIdx < numStates; descStateIdx++){
			sumLeft += leftPartialLikelihoods[descStateIdx * blockDim.x + siteBlockIdx] * 
										((descStateIdx == ancStateIdx) ? diagLeft : offdiagLeft);
                                
			sumRight += rightPartialLikelihoods[descStateIdx * blockDim.x + siteBlockIdx] * 
										((descStateIdx == ancStateIdx) ? diagRight : offdiagRight);
		}

		double prod = sumLeft * sumRight;
		ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx] = prod;

	}
	
	__syncthreads();
	
	if(ancStateIdx == 0 && siteIdx < numSitesWithPadding){
		double maxValue = 0;		
		for(int i = 0; i < numStates; i++){
			double stateValue = ancSequenceSharedRecord[siteBlockIdx * numStates + i];
			if(stateValue > maxValue){
				maxValue = stateValue;
			}
		}
		if(siteIdx < numSites){
			ufScaling[categoryIdx * numSitesWithPadding + siteIdx] *= maxValue;
		}
		
		ufScalingSharedMem[siteBlockIdx] = maxValue;
	}
	__syncthreads();
		
	if(siteIdx < numSitesWithPadding){
		// ... = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]/ufScaling[categoryIdx * numSitesWithPadding + siteIdx]); <- this doesn't work for some reason.
		//Gives NaN as a result when really small numbers are used (1E-37 or smaller).
		if(siteIdx < numSites){
			seqAncNode[ancSequenceIdx] = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]*(1/ufScalingSharedMem[siteBlockIdx]));
		}else{
			seqAncNode[ancSequenceIdx] = 0;
		}
	}
	

}

extern "C"
__global__ void PoissonGpu(int numCategories, int numStates, int numSitesWithPadding, int numSites,
                    double apRate, double* rates, double rateScaling, 
                    float* seqNodeLeft, float* seqNodeRight, float* seqAncNode,
                    double blLeft, double blRight,
                    double* ufScaling){
    
    
    
    //among-site rate heterogenity category index
    int categoryIdx = blockIdx.y;
	
    // sequence site index
    int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	// site index within current block
	int siteBlockIdx = threadIdx.x;
    // index of the ancestral state at the current node
    int ancStateIdx = threadIdx.y;



	__shared__ double ancSequenceSharedRecord[1024];
	__shared__ float leftPartialLikelihoods[1024];
	__shared__ float rightPartialLikelihoods[1024];
	__shared__ double ufScalingSharedMem[512];
	//__shared__ float parentPartialLikelihoods[1024];
	
	// Prefetching partial likelihoods
	if(siteIdx < numSitesWithPadding){
		leftPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeLeft[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
		rightPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeRight[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
	}

    double categoryRate = rates[categoryIdx];
	
	double expLeft = exp(-blLeft * rateScaling * categoryRate * apRate);
    double diagLeft  = 0.05 + (0.95 * expLeft);
    double offdiagLeft = 0.05 - (0.05 * expLeft);

    double expRight = exp(-blRight * rateScaling * categoryRate * apRate);
    double diagRight  = 0.05 + (0.95 * expRight);
    double offdiagRight = 0.05 - (0.05 * expRight);

    int ancSequenceIdx = categoryIdx * numStates * numSitesWithPadding + ancStateIdx * numSitesWithPadding + siteIdx;

	double sumLeft = 0;
	double sumRight = 0;
	if(siteIdx < numSitesWithPadding){

		for(int descStateIdx = 0; descStateIdx < numStates; descStateIdx++){
			sumLeft += leftPartialLikelihoods[descStateIdx * blockDim.x + siteBlockIdx] * 
										((descStateIdx == ancStateIdx) ? diagLeft : offdiagLeft);
                                
			sumRight += rightPartialLikelihoods[descStateIdx * blockDim.x + siteBlockIdx] * 
										((descStateIdx == ancStateIdx) ? diagRight : offdiagRight);
		}

		double prod = sumLeft * sumRight;
		ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx] = prod;

	}
	
	__syncthreads();
	
	if(ancStateIdx == 0 && siteIdx < numSitesWithPadding){
		double maxValue = 0;		
		for(int i = 0; i < numStates; i++){
			double stateValue = ancSequenceSharedRecord[siteBlockIdx * numStates + i];
			if(stateValue > maxValue){
				maxValue = stateValue;
			}
		}
		if(siteIdx < numSites) ufScaling[categoryIdx * numSitesWithPadding + siteIdx] *= maxValue;
		ufScalingSharedMem[siteBlockIdx] = maxValue;
	}
	__syncthreads();
		
	if(siteIdx < numSitesWithPadding){
		// ... = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]/ufScaling[categoryIdx * numSitesWithPadding + siteIdx]); <- this doesn't work for some reason.
		//Gives NaN as a result when really small numbers are used (1E-37 or smaller).
		if(siteIdx < numSites){
			seqAncNode[ancSequenceIdx] = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]*(1/ufScalingSharedMem[siteBlockIdx]));
		}else{
			seqAncNode[ancSequenceIdx] = 0;
		}
	}
	

}


extern "C"
__global__ void KimuraGpu(int numCategories, int numStates, int numSitesWithPadding, int numSites,
                    double kappa, double apRate, double* rates, double rateScaling, 
                    float* seqNodeLeft, float* seqNodeRight, float* seqAncNode,
                    double blLeft, double blRight,
                    double* ufScaling){
    //among-site rate heterogenity category index
    int categoryIdx = blockIdx.y;
	
    // sequence site index
    int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	// site index within current block
	int siteBlockIdx = threadIdx.x;
    // index of the ancestral state at the current node
    int ancStateIdx = threadIdx.y;

	// State indexes
	int A = 0;
	int C = 1;
	int G = 2;
	int T = 3;


	//Shared memory initialization
	__shared__ double ancSequenceSharedRecord[1024];
	__shared__ float leftPartialLikelihoods[1024];
	__shared__ float rightPartialLikelihoods[1024];
	__shared__ double ufScalingSharedMem[512];
	//__shared__ float parentPartialLikelihoods[1024];
	
	// Prefetching partial likelihoods
	if(siteIdx < numSitesWithPadding){
		leftPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeLeft[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
		rightPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeRight[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
	}

    double categoryRate = rates[categoryIdx];
	
	float expLeftA = exp(-(blLeft) * rateScaling * categoryRate * apRate);
	float expLeftB = exp(-(blLeft) * rateScaling * categoryRate * apRate * ((kappa + 1) / 2));
	float diagLeft  = 0.25 + (0.25 * expLeftA) + (0.5 * expLeftB);
	float tiLeft = 0.25 + (0.25 * expLeftA) - (0.5 * expLeftB);
		
	float tvLeft = 0.25 - (0.25 * expLeftA);
		
		
	float expRightA = exp(-(blRight) * rateScaling * categoryRate * apRate);
	float expRightB = exp(-(blRight) * rateScaling * categoryRate * apRate * ((kappa + 1) / 2));
	float diagRight  = 0.25 + (0.25 * expRightA) + (0.5 * expRightB);
	float tiRight = 0.25 + (0.25 * expRightA) - (0.5 * expRightB);
		
	float tvRight = 0.25 - (0.25 * expRightA);

    int ancSequenceIdx = categoryIdx * numStates * numSitesWithPadding + ancStateIdx * numSitesWithPadding + siteIdx;

	double sumLeft = 0;
	double sumRight = 0;
	if(siteIdx < numSitesWithPadding){

		if(ancStateIdx == A){
			/*sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + A * numSites + siteIdx] * diagLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + A * numSites + siteIdx] * diagRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + C * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + C * numSites + siteIdx] * tvRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + G * numSites + siteIdx] * tiLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + G * numSites + siteIdx] * tiRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + T * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + T * numSites + siteIdx] * tvRight;
			//============================================================================================*/
			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * diagLeft;
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * diagRight;
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tvRight;
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tiLeft;
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tiRight;
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tvRight;
			
		}else if(ancStateIdx == C){
			/*sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + A * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + A * numSites + siteIdx] * tvRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + C * numSites + siteIdx] * diagLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + C * numSites + siteIdx] * diagRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + G * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + G * numSites + siteIdx] * tvRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + T * numSites + siteIdx] * tiLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + T * numSites + siteIdx] * tiRight;
			//==============================================================================================*/
			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tvRight;
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * diagLeft;
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * diagRight;
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tvRight;
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tiLeft;
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tiRight;
		}else if(ancStateIdx == G){
			/*sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + A * numSites + siteIdx] * tiLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + A * numSites + siteIdx] * tiRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + C * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + C * numSites + siteIdx] * tvRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + G * numSites + siteIdx] * diagLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + G * numSites + siteIdx] * diagRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + T * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + T * numSites + siteIdx] * tvRight;
			//===============================================================================================*/
			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tiLeft;
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tiRight;
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tvRight;
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * diagLeft;
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * diagRight;
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tvRight;
		}else if(ancStateIdx == T){
			/*sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + A * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + A * numSites + siteIdx] * tvRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + C * numSites + siteIdx] * tiLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + C * numSites + siteIdx] * tiRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + G * numSites + siteIdx] * tvLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + G * numSites + siteIdx] * tvRight;
			sumLeft += seqNodeLeft[categoryIdx * numStates * numSites + T * numSites + siteIdx] * diagLeft;
			sumRight += seqNodeRight[categoryIdx * numStates * numSites + T * numSites + siteIdx] * diagRight;
			//===============================================================================================*/
			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tvRight;
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tiLeft;
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tiRight;
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tvLeft;
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tvRight;
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * diagLeft;
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * diagRight;
		}

		double prod = sumLeft * sumRight;
		ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx] = prod;

	}
	
	__syncthreads();
	
	if(ancStateIdx == 0 && siteIdx < numSitesWithPadding){
		double maxValue = 0;		
		for(int i = 0; i < numStates; i++){
			double stateValue = ancSequenceSharedRecord[siteBlockIdx * numStates + i];
			if(stateValue > maxValue){
				maxValue = stateValue;
			}
		}
		if(siteIdx < numSites) ufScaling[categoryIdx * numSitesWithPadding + siteIdx] *= maxValue;
		ufScalingSharedMem[siteBlockIdx] = maxValue;
	}
	__syncthreads();
		
	if(siteIdx < numSitesWithPadding){
		// ... = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]/ufScaling[categoryIdx * numSitesWithPadding + siteIdx]); <- this doesn't work for some reason.
		//Gives NaN as a result when really small numbers are used (1E-37 or smaller).
		if(siteIdx < numSites){
			seqAncNode[ancSequenceIdx] = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]*(1/ufScalingSharedMem[siteBlockIdx]));
		}else{
			seqAncNode[ancSequenceIdx] = 0;
		}
	}
}

extern "C"
__global__ void HasegawaGPU(int numCategories, int numStates, int numSitesWithPadding, int numSites,
                    double kappa, double apRate, double* rates, double rateScaling, 
                    float* seqNodeLeft, float* seqNodeRight, float* seqAncNode,
					double* PIj, double* equiFreq,
                    double* branchLengths,
                    double* ufScaling){
    
    
    //among-site rate heterogenity category index
    int categoryIdx = blockIdx.y;
	
    // sequence site index
    int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	// site index within current block
	int siteBlockIdx = threadIdx.x;
    // index of the ancestral state at the current node
    int ancStateIdx = threadIdx.y;
	
	// numerical representations of states A, C, G and T.
	int A = 0;
	int C = 1;
	int G = 2;
	int T = 3;
	

	//Shared memory initialization
	__shared__ double ancSequenceSharedRecord[1024];
	__shared__ float leftPartialLikelihoods[1024];
	__shared__ float rightPartialLikelihoods[1024];
	__shared__ double ufScalingSharedMem[512];
	
	__shared__ double expA[8];
	__shared__ double expB[8];
	__shared__ double diag[8];
	__shared__ double ti[8];
	__shared__ double tv[8];
	
	// Prefetching partial likelihoods
	if(siteIdx < numSitesWithPadding){
		leftPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeLeft[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
		rightPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeRight[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
	}

    double categoryRate = rates[categoryIdx];
	
	if(siteBlockIdx == 0){
		for(int branch = 0; branch < 2; branch++){
			expA[branch * numStates + ancStateIdx] = exp(-(branchLengths[branch]) * rateScaling * categoryRate * apRate);
			expB[branch * numStates + ancStateIdx] = exp(-(branchLengths[branch]) * rateScaling * categoryRate * apRate * (1.0 + PIj[ancStateIdx] * (kappa - 1.0)));
			diag[branch * numStates + ancStateIdx] = equiFreq[ancStateIdx] + equiFreq[ancStateIdx]
							* ((1.0/PIj[ancStateIdx])-1.0) * expA[branch * numStates + ancStateIdx] +((PIj[ancStateIdx]-equiFreq[ancStateIdx])/PIj[ancStateIdx])
							* expB[branch * numStates + ancStateIdx];
			ti[branch * numStates + ancStateIdx] = equiFreq[ancStateIdx] + equiFreq[ancStateIdx]
							* ((1.0/PIj[ancStateIdx])-1.0) * expA[branch * numStates + ancStateIdx] - equiFreq[ancStateIdx]/PIj[ancStateIdx]
							* expB[branch * numStates + ancStateIdx];
			tv[branch * numStates + ancStateIdx] =  equiFreq[ancStateIdx] * (1.0 - expA[branch * numStates + ancStateIdx]);
		}
	}
	
	__syncthreads();

    int ancSequenceIdx = categoryIdx * numStates * numSitesWithPadding + ancStateIdx * numSitesWithPadding + siteIdx;

	double sumLeft = 0;
	double sumRight = 0;
	if(siteIdx < numSitesWithPadding){

		if(ancStateIdx == A){

			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * diag[A];
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * diag[numStates + A];
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tv[C];
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tv[numStates + C];
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * ti[G];
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * ti[numStates + G];
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tv[T];
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tv[numStates + T];
		}else if(ancStateIdx == C){

			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tv[A];
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tv[numStates + A];
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * diag[C];
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * diag[numStates + C];
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tv[G];
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tv[numStates + G];
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * ti[T];
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * ti[numStates + T];
		}else if(ancStateIdx == G){

			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * ti[A];
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * ti[numStates + A];
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tv[C];
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * tv[numStates + C];
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * diag[G];
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * diag[numStates + G];
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tv[T];
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * tv[numStates + T];
		}else if(ancStateIdx == T){

			sumLeft += leftPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tv[A];
			sumRight += rightPartialLikelihoods[A * blockDim.x + siteBlockIdx] * tv[numStates + A];
			sumLeft += leftPartialLikelihoods[C * blockDim.x + siteBlockIdx] * ti[C];
			sumRight += rightPartialLikelihoods[C * blockDim.x + siteBlockIdx] * ti[numStates + C];
			sumLeft += leftPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tv[G];
			sumRight += rightPartialLikelihoods[G * blockDim.x + siteBlockIdx] * tv[numStates + G];
			sumLeft += leftPartialLikelihoods[T * blockDim.x + siteBlockIdx] * diag[T];
			sumRight += rightPartialLikelihoods[T * blockDim.x + siteBlockIdx] * diag[numStates + T];
		}

		double prod = sumLeft * sumRight;
		ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx] = prod;
		//debugging[ancSequenceIdx] = prod;
		// debugging[A] = ti[A];
		// debugging[numStates + A] = ti[numStates + A];
		// debugging[C] = ti[C];
		// debugging[numStates + C] = ti[numStates + C];
		// debugging[G] = ti[G];
		// debugging[numStates + G] = ti[numStates + G];
		// debugging[T] = ti[T];
		// debugging[numStates + T] = ti[numStates + T];

	}
	
	__syncthreads();
	
	if(ancStateIdx == 0 && siteIdx < numSitesWithPadding){
		double maxValue = 0;		
		for(int i = 0; i < numStates; i++){
			double stateValue = ancSequenceSharedRecord[siteBlockIdx * numStates + i];
			if(stateValue > maxValue){
				maxValue = stateValue;
			}
		}
		if(siteIdx < numSites) ufScaling[categoryIdx * numSitesWithPadding + siteIdx] *= maxValue;
		ufScalingSharedMem[siteBlockIdx] = maxValue;
	}
	__syncthreads();
		
	if(siteIdx < numSitesWithPadding){
		// ... = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]/ufScaling[categoryIdx * numSitesWithPadding + siteIdx]); <- this doesn't work for some reason.
		//Gives NaN as a result when really small numbers are used (1E-37 or smaller).
		if(siteIdx < numSites){
			seqAncNode[ancSequenceIdx] = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]*(1/ufScalingSharedMem[siteBlockIdx]));
		}else{
			seqAncNode[ancSequenceIdx] = 0;
		}
	}
	

}

extern "C"
__global__ void GTRGPU(int numCategories, int numStates, int numSitesWithPadding, int numSites, 
                    float* seqNodeLeft, float* seqNodeRight, float* seqAncNode,
					double* TPMleft, double* TPMright,
                    double* ufScaling, int sequence_split_offset){
    
    
    //among-site rate heterogenity category index
    int categoryIdx = blockIdx.y;
    // sequence site index
    int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	// site index within current block
	int siteBlockIdx = threadIdx.x;
    // index of the ancestral state at the current node
    int ancStateIdx = threadIdx.y;



	__shared__ double ancSequenceSharedRecord[1024];
	__shared__ float leftPartialLikelihoods[1024];
	__shared__ float rightPartialLikelihoods[1024];
	__shared__ double ufScalingSharedMem[512];
	__shared__ double sMatrixColumnLeft[64];
	__shared__ double sMatrixColumnRight[64];

	
	// Prefetching partial likelihoods
	if(siteIdx < numSitesWithPadding){
		leftPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeLeft[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
		rightPartialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = seqNodeRight[siteIdx + ancStateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
	}

    int ancSequenceIdx = categoryIdx * numStates * numSitesWithPadding + ancStateIdx * numSitesWithPadding + siteIdx;

	double sumLeft = 0;
	double sumRight = 0;
	

	for(int descStateIdx = 0; descStateIdx < numStates; descStateIdx++){
		
		if(threadIdx.x == 0){
			sMatrixColumnLeft[ancStateIdx] = TPMleft[categoryIdx * numStates * numStates + ancStateIdx * numStates + descStateIdx];
		}else if(threadIdx.x == 1){
			sMatrixColumnRight[ancStateIdx] = TPMright[categoryIdx * numStates * numStates + ancStateIdx * numStates + descStateIdx];
		}
		
		__syncthreads();
		
		if(siteIdx < numSites){
			sumLeft += leftPartialLikelihoods[descStateIdx * blockDim.x + siteBlockIdx] * sMatrixColumnLeft[ancStateIdx];
                               
			sumRight += rightPartialLikelihoods[descStateIdx * blockDim.x + siteBlockIdx] * sMatrixColumnRight[ancStateIdx];
		}
		__syncthreads();
	}

	double prod = sumLeft * sumRight;
	ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx] = prod;
	
	__syncthreads();
	
	if(ancStateIdx == 0 && siteIdx < numSitesWithPadding){
	double maxValue = 0;		
	for(int i = 0; i < numStates; i++){
		double stateValue = ancSequenceSharedRecord[siteBlockIdx * numStates + i];
		if(stateValue > maxValue){
			maxValue = stateValue;
		}
	}
	int scaling_split_offset = sequence_split_offset * numCategories;
	
	if(siteIdx < numSites) ufScaling[categoryIdx * numSitesWithPadding + siteIdx + scaling_split_offset] *= maxValue;
	ufScalingSharedMem[siteBlockIdx] = maxValue;
	}
	__syncthreads();
		
	if(siteIdx < numSitesWithPadding){
		// ... = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]/ufScaling[categoryIdx * numSitesWithPadding + siteIdx]); <- this doesn't work for some reason.
		//Gives NaN as a result when really small numbers are used (1E-37 or smaller).
		if(siteIdx < numSites){
			seqAncNode[ancSequenceIdx] = (float)(ancSequenceSharedRecord[siteBlockIdx * numStates + ancStateIdx]*(1/ufScalingSharedMem[siteBlockIdx]));
		}else{
			seqAncNode[ancSequenceIdx] = 0;
		}
		
	}
	

}


extern "C"
__global__ void transMatrixCalc(int n, double* ad, double* bd, double* ed, double* TPMleft, double* TPMright,
									double blLeft, double blRight, double catRate, double apRate, int cat) {
									
	//MAKE IT FOR LEFT AND RIGHT
    __shared__ double as[32][32];
    __shared__ double bs[32][32];
	__shared__ double es[32];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int x = (blockIdx.x * blockDim.x) + tx;
    int y = (blockIdx.y * blockDim.y) + ty;
        
    double vLeft = 0.0;
	double vRight = 0.0;
    
    int yn = y * n;
    int s = (n + 31) / 32;
	//int wholeBlocks = n/32;
    for(int m=0; m<s; m++) {
		
		int m32 = m * 32;

		as[ty][tx] = ad[yn + (m32 + tx)];
		bs[ty][tx] = bd[(m32 + ty) * n + x];
		es[tx] = ed[m32 + tx];

        __syncthreads();

		//skratiti broj iteracija
		for(int i=0; i+m32<n && i<32; i++) {
			double eigenExponent = exp(blLeft * catRate * apRate * es[i]);
			vLeft += as[ty][i] * eigenExponent * bs[i][tx];
			eigenExponent = exp(blRight * catRate * apRate * es[i]);
			vRight += as[ty][i] * eigenExponent * bs[i][tx];
		}

        __syncthreads();
    }
    if(x < n && y < n){
		TPMleft[cat*n*n + yn + x] = vLeft;
		TPMright[cat*n*n + yn + x] = vRight;
		//TPMleft[cat*n*n + yn + x] = cat;
		//TPMright[cat*n*n + yn + x] = catRate;
	}
}

extern "C"
__global__ void initUnderflowScaling(double* ufScaling, double value, int numElements){
	int elementIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if(elementIdx < numElements){
		ufScaling[elementIdx] = (double)1.0;
	}
}

extern "C"
__global__ void reduceStates(int numCategories, int numStates, int numSitesWithPadding, int numSites,
                    double pInv, double* equiFreq,
                    float* sequence, double* ufScaling, int ufScalingOffset){
    

    //among-site rate heterogenity category index
    int categoryIdx = blockIdx.y;
	
    // sequence site index
    int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	// site index within current block
	int siteBlockIdx = threadIdx.x; 
    // index of the ancestral state at the current node
    int stateIdx = threadIdx.y;



	__shared__ float partialLikelihoods[1024];
	__shared__ double sharedDoubleBuffer[1024];
	
	// Prefetching partial likelihoods
	if(siteIdx < numSitesWithPadding){
		partialLikelihoods[threadIdx.x + blockDim.x * threadIdx.y] = sequence[siteIdx + stateIdx*numSitesWithPadding + categoryIdx*numStates*numSitesWithPadding];
	}
	
	
	if(siteIdx < numSites){	
		sharedDoubleBuffer[stateIdx * blockDim.x + siteBlockIdx] = partialLikelihoods[stateIdx * blockDim.x + siteBlockIdx] * equiFreq[stateIdx]
											* ((1.0 - pInv)/(double)(numCategories)) * ufScaling[categoryIdx * numSitesWithPadding + siteIdx + ufScalingOffset];
	}
	
	__syncthreads();
	
	// Reduce states here
	if(stateIdx == 0 && siteIdx < numSites){
		double cellLikelihoodAccumulator = 0;
		for(int st = 0; st < numStates; st++){
			cellLikelihoodAccumulator += sharedDoubleBuffer[st * blockDim.x + siteBlockIdx];
		}
		ufScaling[categoryIdx * numSitesWithPadding + siteIdx + ufScalingOffset] = cellLikelihoodAccumulator;
	}
}

extern "C"
__global__ void reduceCategories(int numCategories, int numSites, int numSitesWithPadding, double pInv, double* ufScaling, double* invSites, int* weights, int split_offset){
	int categoryIdx = threadIdx.y;
	int siteIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int siteBlockIdx = threadIdx.x;
	
	__shared__ double sharedDoubleBuffer[1024];
	
	if(siteIdx < numSites){
		sharedDoubleBuffer[categoryIdx * blockDim.x + siteBlockIdx] = ufScaling[categoryIdx * numSitesWithPadding + siteIdx + (split_offset*numCategories)];
	}
	
	__syncthreads();
	
	if(categoryIdx == 0 && siteIdx < numSites){
		for(int cat = 1; cat < numCategories; cat++){
			sharedDoubleBuffer[siteBlockIdx] += sharedDoubleBuffer[cat * blockDim.x + siteBlockIdx];
		}
		double siteLikelihoodInv = invSites[siteIdx + split_offset] * pInv;
		ufScaling[siteIdx + split_offset] = log(sharedDoubleBuffer[siteBlockIdx] + siteLikelihoodInv) * weights[siteIdx + split_offset];
	}	
}

extern "C"
__global__ void reduceSites(double* g_odata, double* g_idata, int n){
	
	__shared__ double temp[2115];
	int thid = threadIdx.x;
	int offset = 1;
	
	int ai = thid;  
	int bi = thid + (MAX_ELEMENTS_PER_BLOCK/2);  
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);  
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	//Copy data from global memory to shared memory and apply
	// padding for sizes that are not exponents of 2
	
	int blockOffset = MAX_ELEMENTS_PER_BLOCK * blockIdx.x;
	
	if((blockOffset + ai) < n){
		temp[ai + bankOffsetA] = g_idata[blockOffset + ai];
	}else{
		temp[ai + bankOffsetA] = 0;
	}
	
	if((blockOffset + bi) < n){
		temp[bi + bankOffsetB] = g_idata[blockOffset + bi];
	}else{
		temp[bi + bankOffsetB] = 0;
	}
	
	for(int d = MAX_ELEMENTS_PER_BLOCK >> 1; d > 0; d >>= 1){
		__syncthreads();
		if(thid < d){
			int ai = offset * (2 * thid + 1) - 1;
			int bi = offset * (2 * thid + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	
	if(thid == 0){
		g_idata[blockIdx.x] = temp[MAX_ELEMENTS_PER_BLOCK - 1 + CONFLICT_FREE_OFFSET(MAX_ELEMENTS_PER_BLOCK - 1)];
	}	
	__syncthreads();
	if(thid == 0 && blockIdx.x == 0 && gridDim.x == 1) g_odata[0] = g_idata[0];
}