
#include <hip/hip_runtime.h>
#include "math.h"
#define SMALLEST_FLOAT 1.175494351E-38
extern "C"
__global__ void transMatrixCalc(int n, double* ad, double* bd, double* ed, double* cd,
									double bl, double catRate, double apRate, int catNum) {
    __shared__ double as[32][32];
    __shared__ double bs[32][32];
	__shared__ double es[32];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int x = (blockIdx.x * blockDim.x) + tx;
    int y = (blockIdx.y * blockDim.y) + ty;
        
    double v = 0.0;
    
    int yn = y * n;
    int s = (n + 31) / 32;
	//int wholeBlocks = n/32;
    for(int m=0; m<s; m++) {
		
		int m32 = m * 32;

		as[ty][tx] = ad[yn + (m32 + tx)];
		bs[ty][tx] = bd[(m32 + ty) * n + x];
		es[tx] = ed[m32 + tx];

        __syncthreads();

		//skratiti broj iteracija
		for(int i=0; i+m32<n && i<32; i++) {
			double eigenExponent = exp(bl * catRate * apRate * es[i]);
			v += as[ty][i]* eigenExponent * bs[i][tx];
		}

        __syncthreads();
    }
    if(x < n && y < n) cd[catNum*n*n + yn + x] = v;    
}